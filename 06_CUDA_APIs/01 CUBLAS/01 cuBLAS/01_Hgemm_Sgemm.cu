// dedicated for small handwritten matrices
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <time.h>

#define M 512
#define K 512
#define N 512

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUBLAS(call) { \
    hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS error in %s:%d: %d\n", __FILE__, __LINE__, status); \
        exit(EXIT_FAILURE); \
    } \
}

#undef PRINT_MATRIX
#define PRINT_MATRIX(mat, rows, cols) \
    for (int i = 0; i < rows; i++) { \
        for (int j = 0; j < cols; j++) \
            printf("%8.3f ", mat[i * cols + j]); \
        printf("\n"); \
    } \
    printf("\n");

void cpu_matmul(float *A, float *B, float *C) {
    for (int i = 0; i < M; i++)
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++)
                sum += A[i * K + k] * B[k * N + j];
            C[i * N + j] = sum;
        }
}

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

void init_vector(float *vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = (float)rand() / RAND_MAX;
    }
}

int main() {

    float A[M * K];
    float B[K * N];
    float C_cpu[M * N], C_cublas_s[M * N], C_cublas_h[M * N];
    
    srand(time(NULL));
    init_vector(A, M*K);
    init_vector(B, K*N);

    double cpu_time = 0.0;
    // CPU matmul
    double start_time = get_time();
    cpu_matmul(A, B, C_cpu);
    double end_time = get_time();
    cpu_time += end_time - start_time;

    printf("cpu matmul time: %f\n", cpu_time*1000);

    hipEvent_t start, end;

    hipEventCreate(&start);
    hipEventCreate(&end);

    // CUDA setup
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, M * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, M * N * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice));

    // row major A = 
    // 1.0 2.0 3.0 4.0
    // 5.0 6.0 7.0 8.0

    // col major A = 
    // 1.0 5.0
    // 2.0 6.0
    // 3.0 7.0
    // 4.0 8.0

    // memory layout (row)
    // 1.0 2.0 3.0 4.0 5.0 6.0 7.0 8.0

    // memory layout (col)
    // 1.0 5.0 2.0 6.0 3.0 7.0 4.0 8.0
    
    // cuBLAS SGEMM
    hipStream_t s1, s2;
    hipStreamCreate(&s1);
    hipblasSetStream(handle, s1);
    float alpha = 1.0f, beta = 0.0f;
    hipEventRecord(start, s1);
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N));
    hipEventRecord(end, s1);
    float ms = 0;
    hipEventSynchronize(end);
    hipEventElapsedTime(&ms, start, end);
    printf("FP32 gpu matmul time: %f\n", ms);
    CHECK_CUDA(hipMemcpy(C_cublas_s, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));
    

    // cuBLAS HGEMM
    half *d_A_h, *d_B_h, *d_C_h;
    CHECK_CUDA(hipMalloc(&d_A_h, M * K * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_B_h, K * N * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_C_h, M * N * sizeof(half)));

    // Convert to half precision on CPU
    half A_h[M * K], B_h[K * N];
    for (int i = 0; i < M * K; i++) {
        A_h[i] = __float2half(A[i]);
    }
    for (int i = 0; i < K * N; i++) {
        B_h[i] = __float2half(B[i]);
    }

    // Copy half precision data to device
    CHECK_CUDA(hipMemcpy(d_A_h, A_h, M * K * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B_h, B_h, K * N * sizeof(half), hipMemcpyHostToDevice));

    // Copy result back to host and convert to float
    
    hipblasHandle_t handle2;
    CHECK_CUBLAS(hipblasCreate(&handle2));

    hipEvent_t start2, end2;

    hipEventCreate(&start2);
    hipEventCreate(&end2);
    
    half C_h[M * N];
    hipStreamCreate(&s2);
    hipblasSetStream(handle2, s2);
    hipEventRecord(start2, s2);
    CHECK_CUBLAS(hipblasGemmEx(
        handle2,
        HIPBLAS_OP_T, HIPBLAS_OP_T,
        N, M, K,
        &alpha,
        d_B_h, HIP_R_16F, N,
        d_A_h, HIP_R_16F, K,
        &beta,
        d_C_h, HIP_R_16F, N,
        HIP_R_16F,
        CUBLAS_GEMM_DFALT_TENSOR_OP
    ));
    
    hipEventRecord(end2, s2);
    hipEventSynchronize(end2);
    hipEventElapsedTime(&ms, start2, end2);
    CHECK_CUDA(hipMemcpy(C_h, d_C_h, M * N * sizeof(half), hipMemcpyDeviceToHost));

    printf("FP16 gpu matmul time: %f\n", ms);
    for (int i = 0; i < M * N; i++) {
        C_cublas_h[i] = __half2float(C_h[i]);
    }

    // Print results
    // printf("Matrix A (%dx%d):\n", M, K);
    // PRINT_MATRIX(A, M, K);
    // printf("Matrix B (%dx%d):\n", K, N);
    // PRINT_MATRIX(B, K, N);
    // printf("CPU Result (%dx%d):\n", M, N);
    // PRINT_MATRIX(C_cpu, M, N);
    // printf("cuBLAS SGEMM Result (%dx%d):\n", M, N);
    // PRINT_MATRIX(C_cublas_s, M, N);
    // printf("cuBLAS HGEMM Result (%dx%d):\n", M, N);
    // PRINT_MATRIX(C_cublas_h, M, N);

    // Clean up
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(d_A_h));
    CHECK_CUDA(hipFree(d_B_h));
    CHECK_CUDA(hipFree(d_C_h));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(end));
    CHECK_CUDA(hipEventDestroy(start2));
    CHECK_CUDA(hipEventDestroy(end2));
    CHECK_CUDA(hipStreamDestroy(s1));
    CHECK_CUDA(hipStreamDestroy(s2));

    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUBLAS(hipblasDestroy(handle2));

    return 0;
}